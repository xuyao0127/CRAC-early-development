
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

int main()
{
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    sleep(10);
    return 0;
}
