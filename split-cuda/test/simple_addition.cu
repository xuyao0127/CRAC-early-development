
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
/*
extern "C"
{
 void _ZL24__sti____cudaRegisterAllv(){
 }
} */

__global__ void add(int a, int b, int *c)
{
	*c = a+b;
	printf("Inside %d + %d = %d\n", a, b, *c);
}

int main(int argc, char **argv)
{
	// test
	int a = 3, b = 3, c=0;
	int *cuda_c = NULL;
	printf("before any cuda call\n");
	printf("Hey\n");
	fflush(stdout);
	hipMallocManaged(&cuda_c, sizeof(int));
        printf("\nckpt point 1: before kernel\n");
        fflush(stdout);
        sleep(10);
	add<<<1,1>>>(a, b, cuda_c);
	hipDeviceSynchronize();
        printf("ckpt point 2: after kernel\n");
        fflush(stdout);
        sleep(10);
	hipError_t ret = hipMemcpy(&c, cuda_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("device 1: %d \n", *cuda_c);
        printf("ckpt point 3: after copying to host\n");
        fflush(stdout);
        sleep(10);

	hipSetDevice(1);
	int *cuda_c2 = NULL;
	hipMallocManaged(&cuda_c2, sizeof(int));
	//cudaSetDevice(0);
	printf("device 1: %d \n", *cuda_c);
	ret = hipMemcpy(cuda_c2, cuda_c, sizeof(int), hipMemcpyDeviceToDevice);
	printf("error: %s \n", hipGetErrorString(ret));	
        printf("device 1: %d \n", *cuda_c);
        printf("device 2: %d \n", *cuda_c2);

	hipDeviceSynchronize();
	ret = hipMemcpy(&c, cuda_c2, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	//printf("error: %s \n", cudaGetErrorString(ret));
	printf("device 1: %d \n", *cuda_c);
	hipDeviceSynchronize();
	printf("device 2: %d \n", *cuda_c2);
	printf("host %d \n", c);
	fflush(stdout);
	exit(EXIT_SUCCESS);
}
